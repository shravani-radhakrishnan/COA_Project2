#include "hip/hip_runtime.h"
/*
 * _MATRIXMUL_GPU_CU_
 *
 * 2022 Mert SIDE
 *
 * CS5375 Computer Systems Organization and Architecture
 * Guest Lecture: GPU Programming
 *
 * Multiplying two matrices on the GPU
 *
 */

#include <iostream>
#include <stdio.h>
#include <stdlib.h>

// ------------------------------------------------------------------ GPUmatmul
__global__
void GPUmatmul(int N, double *x, double *y, double *ans)
{
  // Compute each thread's global row and column index
  int row_num = (blockIdx.y * blockDim.y) + threadIdx.y;
  int col_num = (blockIdx.x * blockDim.x )+ threadIdx.x;

  // Iterate over row, and down column
  ans[row_num * N + col_num] = 0;
  for (int k = 0; k < N; k++)
  {
    // Accumulate results for a single element
    ans[row_num * N + col_num] += x[row_num * N + k] * y[k * N + col_num];
  }
}

// ---------------------------------------------------------------------- check
bool check(int N, double *ans)
{
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      if (ans[i * N + j] != 20.0)
        return false;
    }
  }
  return true;
}

// ----------------------------------------------------------------------- MAIN
int main(void)
{
  // size of matrix
  int N = 1 << 9; // binary left-shift: 1 * 2^9 = 512
  printf("Size of matrix (N) is %d by %d.\n", N, N);
  int iter = 3;
  clock_t t;

  // Martices
  double *x, *y, *ans;

  // TODO: Allocate Unified Memory - accessible from both CPU and GPU
// ...
  // ...
  // ...

  // ..........................................................................
  // initialize x,y and ans arrays on the host
  hipMallocManaged((void **)&x, sizeof(float) * N * N);
  hipMallocManaged((void **)&y, sizeof(float) * N * N);
  hipMallocManaged((void **)&ans, sizeof(float) * N * N);

  int device = -1;
  hipMemPrefetchAsync(x, sizeof(float) * N * N, device, NULL);
  hipMemPrefetchAsync(y, sizeof(float) * N * N, device, NULL);
  hipMemPrefetchAsync(ans, sizeof(float) * N * N, device, NULL);
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      x[i * N + j] = 5;
      y[i * N + j] = (i == j ? 1 : 0);
      ans[i * N + j] = (double)0.000000000000;
    }
  }

  int blockSize = 16; // 16*16 = 256
  int gridSize = (int)(N / blockSize);
  // ..........................................................................
  double avg = 0;
  std::cout << "Starting optimized GPU computation" << std::endl;

  dim3 grid(gridSize, gridSize);
  dim3 threads(blockSize, blockSize);
  // Run kernel on GPU
  for (int i = 0; i <= iter; i++)
  {
    t = clock();
    GPUmatmul<<<grid, threads>>>(N, x, y, ans);
    hipDeviceSynchronize();
    t = clock() - t;
    if (i)
      avg += t; // we will ignore the first run
    // printf ("It took GPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
  }

  avg = t;
  avg /= iter;
  avg /= CLOCKS_PER_SEC;
  avg *= 1000;
  printf("It took %lf ms on avg.\n", avg);
  if (check(N, ans))
    std::cout << "RUN OK." << std::endl;
  else
    std::cout << "RUN OK." << std::endl;

  // ..........................................................................

  // TODO: Free memory
  // ...
  // ...
  // ...
  hipFree(x);
  hipFree(y);
  hipFree(ans);

  return 0;
}
/* EOF */
                                                                                                                                                   2,1           Top
